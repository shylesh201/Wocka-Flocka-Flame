
#include <hip/hip_runtime.h>
#include<stdio.h>
#define n 8
__global__ void sort(int *a)
{
    int i=threadIdx.x;
  for(int j=0;j<n;j++)
  {
    if(j%2==0 && (i*2+1)<n)
    {
      if(a[2*i]>a[2*i+1])
      {
        int temp=a[2*i];
        a[2*i]=a[2*i+1];
        a[2*i+1]=temp;
      }
    }
    if(j%2==1 && (2*i+2)<n)
    {
      if(a[2*i+1]>a[2*i+2])
      {
        int temp=a[2*i+1];
        a[2*i+1]=a[2*i+2];
        a[2*i+2]=temp;
      }
    }
  }
}
int main()
{
  int a[n]={2,1,4,9,5,3,6,10};
  int *da;
  hipMalloc((void**)&da,n*sizeof(int));
  hipMemcpy(da,&a,n*sizeof(int),hipMemcpyHostToDevice);
  sort<<<1,n>>>(da);
  hipMemcpy(&a,da,n*sizeof(int),hipMemcpyDeviceToHost);
  printf("Sorted Order: ");
  for(int i=0;i<n;i++)
    printf("%d ",a[i]);
  return 0;
}