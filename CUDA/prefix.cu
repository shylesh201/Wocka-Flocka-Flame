#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void prefixsum(int *a,int n,double logn)
{
    int i=threadIdx.x;
    for (int j=0;j<logn;j++)
    {   int cd = i-pow(2,j);
        if(i>=pow(2,j))
        {   
            a[i]=a[i]+a[cd];
        }
    }
}

int main()
{
    int n;
    printf("Enter the number of elements in the array:");
    scanf("%d",&n);
    int a[n];
    printf("Enter the elements of the array:\n");
    for(int i=0;i<n;i++)
    {
        scanf("%d",&a[i]);
    }
    int *d_a;
    double x = log(n)/log(2);
    hipMalloc((void **)&d_a,n*sizeof(int));
    hipMemcpy(d_a,a,n*sizeof(int),hipMemcpyHostToDevice);
    prefixsum<<<1,n>>>(d_a,n,x);
    hipMemcpy(a,d_a,n*sizeof(int),hipMemcpyDeviceToHost);
    printf("The prefix sum of the array is:");
    for(int i=0;i<n;i++)
    {
        printf("%d ",a[i]);
    }
    printf("\n");
    hipFree(d_a);
    return 0;
}