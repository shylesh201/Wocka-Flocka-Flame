
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 8
__device__ struct point {
        int x;
        int y;
};

__device__ struct point succ[N][N];
__device__ int position[N][N];

__global__ void preorder(int *parent, int *sibling, int *child, int *adj, int *preo) {
        int i = threadIdx.x;
        int j = threadIdx.y;
        int gind = j*N+i;
        if(adj[gind]==1) {
                 printf("Edge (%d, %d)\n",i, j);
                if(parent[i] == j) {
                        if(sibling[i]!=(-1)) {
                                struct point pt;
                                pt.x = j;
                                pt.y = sibling[i];
                                succ[i][j] = pt;
                        }
                        else if(parent[j]!=(-1)) {
                                 struct point pt;
                                pt.x = j;
                                pt.y = parent[j];
                                succ[i][j] = pt;
                        }
                        else {
                                 struct point pt;
                                pt.x = i;
                                pt.y = j;
                                succ[i][j] = pt;
                                preo[j] = 1;
                        }
                }
                else {
                        if(child[j]!=(-1)) {
                                 struct point pt;
                                pt.x = j;
                                pt.y = child[j];
                                succ[i][j] = pt;
                        }
                        else {
                                 struct point pt;
                                pt.x = j;
                                pt.y = i;
                                succ[i][j] = pt;
                        }
			}
                __syncthreads();
                if(parent[i]==j) position[i][j] = 0;
                else position[i][j] = 1;
                int logval = (int)ceil(log2((double)(2*(N-1))));
                printf("Successor of (%d, %d) = (%d, %d)\n",i, j, succ[i][j].x, succ[i][j].y);

                for(int k=1; k<=logval; k++) {
                        __syncthreads();
                        struct point pt = succ[i][j];
                        position[i][j] = position[i][j]+position[pt.x][pt.y];
                        succ[i][j] = succ[pt.x][pt.y];
                }
                if(i==parent[j]) preo[j] = N+1-position[i][j];
                __syncthreads();
        }

}

int main() {
        int parents[] = {-1, 0, 0, 1, 1, 2, 4, 4};
        int sibling[] = {-1, 2, -1, 4, -1, -1, 7, -1};
        int children[] = {1, 3, 5, -1, 6, -1, -1, -1};
        int *parent, *sib, *child, *preo, *ordered, *adj;
        ordered = (int *)malloc(sizeof(int)*N);
        hipMalloc((int **)&parent, sizeof(int)*N);
        hipMalloc((int **)&sib, sizeof(int)*N);
        hipMalloc((int **)&child, sizeof(int)*N);
        hipMalloc((int **)&preo, sizeof(int)*N);
        hipMalloc((int **)&adj, sizeof(int)*N*N);
        int adjacency[N][N];
        memset(adjacency, 0, sizeof(adjacency));
        for(int i=0; i<N; i++) {
                for(int j=0; j<N; j++) {
                        if(parents[j]!=-1 && parents[j]==i) {
                                adjacency[i][j] = 1;
                                adjacency[j][i] = 1;
                        }
                }
        }
        hipMemcpy(parent, parents, sizeof(int)*N, hipMemcpyHostToDevice);
        hipMemcpy(sib, sibling, sizeof(int)*N, hipMemcpyHostToDevice);
        hipMemcpy(child, children, sizeof(int)*N, hipMemcpyHostToDevice);
        hipMemcpy(adj, adjacency, sizeof(int)*N*N, hipMemcpyHostToDevice);
        dim3 grid(1);
        dim3 block(N, N);
        preorder<<<grid, block>>>(parent, sib, child, adj, preo);
        hipMemcpy(ordered, preo, sizeof(int)*N, hipMemcpyDeviceToHost);
        int preordered[N];
        for(int i=0; i<N; i++) {
                preordered[ordered[i]-1] = i;
        }
        for(int i=0; i<N; i++) {
                printf("%d ", preordered[i]);
        }
        printf("\n");
        free(ordered);
        hipFree(parent);
        hipFree(sib);
        hipFree(child);
        hipDeviceReset();
}
