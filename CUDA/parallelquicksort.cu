
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>

#define N 20

void initialize(int *a) {
        for(int i = 0; i < N; i++) {
                a[i] = rand() % (100 - 10 + 1) + 10;
        }
}

__device__ int d_size;

__global__ void partition(int *arr, int *lstack, int *hstack, int n) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        d_size = 0;
        __syncthreads();
        if(idx < n) {
                int h = hstack[idx], l = lstack[idx], x = arr[h], i = l - 1;
                int temp;
                for(int j = l; j < h; j++) {
                        if(arr[j] <= x) {
                                i++;
                                temp = arr[i];
                                arr[i] = arr[j];
                                arr[j] = temp;
                        }
                }
                temp = arr[i + 1];
                arr[i + 1] = arr[h];
                arr[h] = temp;
                int p = i + 1;
                if(p - 1 > l) {
                        int ind = atomicAdd(&d_size, 1);
                        lstack[ind] = l;
                        hstack[ind] = p - 1;
                }
                if(p + 1 < h) {
                        int ind = atomicAdd(&d_size, 1);
                        lstack[ind] = p + 1;
                        hstack[ind] = h;
                }
        }
}

void quickSort(int *arr) {
        int low = 0, high = N - 1;
        int lstack[high - low + 1], hstack[high - low + 1];
        int top = -1, *da, *dl, *dh, size = (high - low + 1) * sizeof(int);
        lstack[++top] = low;
        hstack[top] = high;

        hipMalloc(&da, size);
        hipMemcpy(da, arr, size, hipMemcpyHostToDevice);

        hipMalloc(&dl, size);
        hipMemcpy(dl, lstack, size, hipMemcpyHostToDevice);

        hipMalloc(&dh, size);
        hipMemcpy(dh, hstack, size, hipMemcpyHostToDevice);

        int nt, nb, ni;
        nt = nb = ni = 1;

        while(ni > 0) {
                partition<<<nb, nt>>>(da, dl, dh, ni);
                int ans;
                hipMemcpyFromSymbol(&ans, HIP_SYMBOL(d_size), sizeof(int),0, hipMemcpyDeviceToHost);
                if(ans < N * nt) {
                        nt = ans;
                }
                else {
                        nt = N * nt;
                        nb = ans / nt + (ans % nt == 0 ? 0 : 1);
                }
                ni = ans;
                hipMemcpy(arr, da, (high - low + 1) * sizeof(int), hipMemcpyDeviceToHost);
        }
}

int main() {
        int *a = (int* )malloc(N * sizeof(int));
        initialize(a);
        quickSort(a);
        for(int i = 0; i < N; i++) {
                printf("%d ", a[i]);
        }
        printf("\n");
}

