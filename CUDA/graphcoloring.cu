#include<hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>

__global__ void graph(int *a,int *b,int *c)
{
        int m=threadIdx.x;
        int t=3*threadIdx.x;
        if((a[t]==a[t+1]) || (a[t+1]==a[t+2]))
                b[m]=0;
        atomicAdd(c,b[m]);
}

int main(){
        int i,b[8],c;
        int *da,*db,*dc;
        int a[8][3] = {{0,0,0},{0,0,1},{0,1,0},{0,1,1},{1,0,0},{1,0,1},{1,1,0},{1,1,1}};
        for(i=0;i<8;i++)
        {
                b[i]=1;
        }
        hipMalloc((void**)&da,24*sizeof(int));
        hipMalloc((void**)&db,8*sizeof(int));
        hipMalloc((void**)&dc,sizeof(int));
        hipMemcpy(da,&a,24*sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(db,&b,8*sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(dc,&c,sizeof(int),hipMemcpyHostToDevice);
        graph<<<1,8>>>(da,db,dc);
        hipMemcpy(&b,db,8*sizeof(int),hipMemcpyDeviceToHost);
        hipMemcpy(&c,dc,sizeof(int),hipMemcpyDeviceToHost);
        printf("Possible Combinations are:\n");
        for(i=0;i<8;i++)
        {
                if(b[i]==1)
                        printf("%d%d%d\n",a[i][0],a[i][1],a[i][2]);
        }
        printf("Number of combinatons:%d\n",c);
        return 0;
}
