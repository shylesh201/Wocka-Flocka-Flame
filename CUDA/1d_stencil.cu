#include <hip/hip_runtime.h>
#include<stdio.h>
#include<sys/time.h>

#define RADIUS 4
#define BDIM 8

// constant memory
__constant__ float coef[RADIUS + 1];

/*
// FD coeffecient
#define a0     0.00000f
#define a1     0.80000f
#define a2    -0.20000f
#define a3     0.03809f
#define a4    -0.00357f
*/

#define a0 0
#define a1 1
#define a2 2
#define a3 3
#define a4 4

double cpuSecond(){
        struct timeval tp;
        gettimeofday(&tp, NULL);
        return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

void initialData(float *in,  const int size)
{
    for (int i = 0; i < size; i++)
    {
//        in[i] = (float)(rand() & 0xFF) / 100.0f;
in[i]=i+1;
    }
}

void printData(float *in,  const int size)
{
    for (int i = RADIUS; i < size; i++)
    {
        printf("%f ", in[i]);
    }

    printf("\n");
}

void cpu_stencil_1d (float *in, float *out, int isize)
{
    for (int i = RADIUS; i <= isize; i++)
    {
        float tmp = a1 * (in[i + 1] - in[i - 1])
                    + a2 * (in[i + 2] - in[i - 2])
                    + a3 * (in[i + 3] - in[i - 3])
                    + a4 * (in[i + 4] - in[i - 4]);
        out[i] = tmp;
    }
}


__global__ void stencil_1d(float *in, float *out, int N)
{
    // shared memory
    __shared__ float smem[BDIM + 2 * RADIUS];

    // index to global memory
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

        // index to shared memory for stencil calculatioin
        int sidx = threadIdx.x + RADIUS;

        // Read data from global memory into shared memory
        smem[sidx] = in[idx];
printf("\nsmem[%d]=in[%d] by %d, value is %f",sidx,idx,threadIdx.x,in[idx]);
__syncthreads();
        // read halo part to shared memory
        if (threadIdx.x < RADIUS)
        {
            smem[sidx - RADIUS] = in[idx - RADIUS];
            smem[sidx + BDIM] = in[idx + BDIM];
printf("\nsmem[%d]=in[%d] by %d, value is %f",sidx-RADIUS,idx-RADIUS,threadIdx.x,in[idx-RADIUS]);
printf("\nsmem[%d]=in[%d] by %d,value is %f",sidx+BDIM,idx+BDIM,threadIdx.x,in[idx+BDIM]);
        }

        // Synchronize (ensure all the data is available)
        __syncthreads();

        // Apply the stencil
        float tmp = 0.0f;
#pragma unroll
        for (int i = 1; i <= RADIUS; i++)
        {
            tmp += coef[i] * (smem[sidx + i] - smem[sidx - i]);
        }

        // Store the result
        out[idx] = tmp;
printf("\nin[%d] is %f",idx,in[threadIdx.x]);
printf("\nout[%d] = %f by %d", idx,tmp,threadIdx.x);
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("%s starting transpose at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    hipSetDevice(dev);

    // set up data size
    int isize = 1 << 3;

    size_t nBytes = (isize + 2 * RADIUS) * sizeof(float);
    printf("array size: %d ", isize);

    bool iprint = 1;

    // allocate host memory
    float *h_in    = (float *)malloc(nBytes);
    float *hostRef = (float *)malloc(nBytes);
    float *gpuRef  = (float *)malloc(nBytes);

    // allocate device memory
    float *d_in, *d_out;
    hipMalloc((float**)&d_in, nBytes);
    hipMalloc((float**)&d_out, nBytes);

    // initialize host array
    initialData(h_in, isize + 2 * RADIUS);

   // Copy to device
    hipMemcpy(d_in, h_in, nBytes, hipMemcpyHostToDevice);

    // set up constant memory
    const float h_coef[] = {a0, a1, a2, a3, a4};
    hipMemcpyToSymbol( HIP_SYMBOL(coef), h_coef, (RADIUS + 1) * sizeof(float));

    // launch configuration
    hipDeviceProp_t info;
    hipGetDeviceProperties(&info, 0);
    dim3 block(BDIM, 1);
    dim3 grid(info.maxGridSize[0] < isize / block.x ? info.maxGridSize[0] :
            isize / block.x, 1);
    printf("(grid, block) %d,%d \n ", grid.x, block.x);
    double istart = cpuSecond();
    // Launch stencil_1d() kernel on GPU
    stencil_1d<<<1, 8>>>(d_in + RADIUS, d_out + RADIUS, isize);
    double ielapsed = cpuSecond() - istart;
    // Copy result back to host
    hipMemcpy(gpuRef, d_out, nBytes, hipMemcpyDeviceToHost);

    // apply cpu stencil
    double cpustart = cpuSecond();
    cpu_stencil_1d(h_in, hostRef, isize);
    double cpuelapsed = cpuSecond() - cpustart;
    // print out results
    if(iprint)
    { printf("\nisize is %d\n",isize);
        printData(gpuRef, isize);
    //    printData(hostRef, isize);
    }
    printf("GPU Elapsed Time %lf\n",ielapsed);
    printf("CPU Elapsed Time %lf\n",cpuelapsed);
    // Cleanup
    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    free(hostRef);
    free(gpuRef);

    // reset device
    hipDeviceReset();
    return EXIT_SUCCESS;
}
