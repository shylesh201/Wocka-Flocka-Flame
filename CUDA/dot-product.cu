#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void dot(int *a, int *b, int *c, int nx, int ny)
{
  int k = threadIdx.x;
  c[k] = a[k] * b[k];
}

__global__ void red(int * c)
{
  int t = threadIdx.x;
  printf("%d",blockDim.x);
  for(int a = 1; a < blockDim.x; a *= 2)
  {
    if(t % ( 2*a) == 0 && t+a < blockDim.x)
    {
      c[t] += c[t+a];
    }
  }
}

int main()
{
  int a[4] = {1, 2, 3, 4};
  int b[4] = {1, 2, 3, 4};
  int c[4] = {0, 0, 0, 0};

  int *da, *db, *dc, size = 4*sizeof(int);
  hipMalloc((void **) & da, size);
  hipMemcpy(da, a, size, hipMemcpyHostToDevice);
  hipMalloc((void **)&db, size);
  hipMemcpy(db, b, size, hipMemcpyHostToDevice);
  hipMalloc((void **)&dc, size);
  hipMemcpy(dc, c, size, hipMemcpyHostToDevice);
  dot<<<1,4>>>(da, db, dc, 1, 4);
  red<<<1,4>>>(dc);
  hipMemcpy(&c, dc, size, hipMemcpyDeviceToHost);
  printf("%d", c[0]);
  return 0;
}
    