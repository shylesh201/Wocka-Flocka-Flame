
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

__global__ void sum(int *a, int n)
{
int i = threadIdx.x + blockIdx.x * blockDim.x;
for(int j=0; j<=3; j++)
{
if( ( (i % (int)pow(2, j)) == 0 ) && ( (2*i + (int) pow(2, j)) < n) )
{
a[2*i] = a[2*i] + a[2*i + (int)pow(2,j)];
}
}
if(i==0)
printf("###%d",a[0]);
}

int main()
{
int n = 10;
int size = n * sizeof(int);
int *ha;
ha = (int *)malloc(size);

for(int i=1; i<=n; i++)
ha[i-1] = i;

int *da;
hipMalloc(&da, size);
                           
hipMemcpy(da, ha, size, hipMemcpyHostToDevice);

sum<<<1, n/2>>>(da, n);
hipDeviceSynchronize();

//int s;
//cudaMemcpy(s, da[0], sizeof(int), cudaMemcpyDeviceToHost);

//printf("%d", s);
return 0;
}
