#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>

__constant__ int key;

__global__ void linsearch(int *a,int *p){
    int t = blockDim.x*blockIdx.x + threadIdx.x;
    if(a[t] == key){
        *p = t;
    }
}
int main(){
    int n,k,*darr,*p,pos=-1;
    n = 10;
    int arr[n] = {1,2,3,4,5,6,7,8,9,0};
    k = 7;
    hipMalloc((void**)&darr,n*sizeof(int));
    hipMemcpy(darr,&arr,n*sizeof(int),hipMemcpyHostToDevice);
    hipMalloc((void**)&p,sizeof(int));
    hipMemcpy(p,&pos,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(key),&k,sizeof(int));
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    linsearch<<<2,5>>>(darr,p);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipMemcpy(&pos,p,sizeof(int),hipMemcpyDeviceToHost);
    printf("Element Found At : %d\n",pos);
    hipFree(p);
    hipFree(darr);
    return 0;
}
