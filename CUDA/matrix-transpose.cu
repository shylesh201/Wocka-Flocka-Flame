#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<sys/time.h>

#define N 4
#define M 4
#define BDIMX 2
#define BDIMY 2

__global__ void transpose(int *a, int *b) {
        __shared__ int temp[BDIMY][BDIMX];
        int ix = threadIdx.x + blockIdx.x * blockDim.x;
        int iy = threadIdx.y + blockIdx.y * blockDim.y;
        int ti = iy * N + ix;
        int bidx = threadIdx.x + threadIdx.y * blockDim.x;
        int irow = bidx / blockDim.y;
        int icol = bidx % blockDim.y;
        ix = icol + blockIdx.y * blockDim.y;
        iy = irow + blockIdx.x * blockDim.x;
        int to = iy * M + ix;
        if(ix < N && iy < M) {
                temp[threadIdx.y][threadIdx.x] = a[ti];
                __syncthreads();
                b[to] = temp[icol][irow];
        }
}

double cpuSecond() {
        struct timeval tp;
        gettimeofday(&tp, NULL);
        return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

int main() {
        int *a, *b;
        int size = N * M * sizeof(int);
        a = (int* )malloc(size);
        b = (int* )malloc(size);
        for(int i = 0; i < N * M; i++) {
                a[i] = i;
        }
        printf("Initial Array: \n");
        for(int i = 0; i < N; i++) {
                for(int j = 0; j < M; j++) {
                        printf("%d ", a[i * M + j]);
                }
                printf("\n");
        }

        int *da, *db;
        hipMalloc((void** )&da, size);
        hipMalloc((void** )&db, size);
        hipMemcpy(da, a, size, hipMemcpyHostToDevice);
        dim3 block(BDIMX, BDIMY);
        dim3 grid(2, 2);
        double istart = cpuSecond();
        transpose<<<grid, block>>>(da, db);
        hipDeviceSynchronize();
        double ielapsed = cpuSecond() - istart;
        hipMemcpy(b, db, size, hipMemcpyDeviceToHost);
        printf("Final Array: \n");
        for(int i = 0; i < N; i++) {
                for(int j = 0; j < M; j++) {
                        printf("%d ", b[i * M + j]);
                }
                printf("\n");
        }
        printf("Elapsed Time : %lf\n", ielapsed);
}
