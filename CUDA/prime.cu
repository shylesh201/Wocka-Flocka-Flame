
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void find_prime(int *a, int x, int y)
{
int i =  threadIdx.x + blockIdx.x * blockDim.x;
if(i<=y-x)
{
int count = 0;
for(int j=2; j<=a[i]; j++)
{       
if(a[i] % j == 0)
{
count = count + 1;
}
}
if(count == 1)
{
;
}
else
{
a[i] = -1;
}
}
}

void init_array(int *A, int x, int y)
{
for(int i=0; i<=y-x; i++)
{
A[i] = x+i;
}
}

int main()
{
int a, b;
int *ha;

printf("a: ");
scanf("%d", &a);
printf("\nb: ");
scanf("%d", &b);

int n = b-a+1;
int size = n * sizeof(int);
ha = (int *)malloc(size);

init_array(ha, a, b);
//for(int i=0; i<=b-a; i++)
//printf("%d ", ha[i]);
//printf("\n");

int *da;
//cudaMalloc((void **) &da, size);
hipMalloc(&da, size);
hipMemcpy(da, ha, size, hipMemcpyHostToDevice);
find_prime<<<1, b-a+1>>>(da, a, b);
//cudaDeviceSynchronize();
int *hb;
hb = (int *)malloc(size);

hipMemcpy(hb, da, size, hipMemcpyDeviceToHost);

for(int i=0; i<=b-a; i++)
{
if(hb[i] != -1)
{
printf("%d ", hb[i]);
}
}
printf("\n");
return 0;
}
