#include<hip/hip_runtime.h>
#include<stdio.h>

__global__ void pararedn(int* a)
{
  int t = threadIdx.x;
  for(int j = 1; j < blockDim.x; j*=2)
  {
    if(t % (2*j) == 0 && (t+j) < blockDim.x)
    {
      a[t] = a[t] + a[t+j];
    }
    //printf("%d ", a[t]);
  }
}

int main() 
{
  int arr[10];
  for(int i = 0; i < 10; i++) 
  {
    arr[i] = i+1;
  }
  for(int i = 0; i < 10; i++)
  {
    printf("%d ", arr[i]);
  }
  int *in;
  hipMalloc((void**)&in, 10*sizeof(int));
  hipMemcpy(in, &arr, 10 * sizeof(int), hipMemcpyHostToDevice);
  //cudaMemcpy(out, &sol, 6 * sizeof(int), cudaMemcpyHostToDevice);
  printf("before call\n");
  pararedn<<<1, 10>>>(in);
  printf("after call\n");
  hipMemcpy(&arr, in, 10*sizeof(int), hipMemcpyDeviceToHost);
  printf("%d \n", arr[0]);
  hipFree(in);
  //cudaFree(out);
}