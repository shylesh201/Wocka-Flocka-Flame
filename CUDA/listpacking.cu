#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include <ctype.h>

__global__ void prefixsum(int *a,int n,double logn)
{
    int i=threadIdx.x;
    for (int j=0;j<logn;j++)
    {   int cd = i-pow(2,j);
        if(i>=pow(2,j))
        {   
            a[i]=a[i]+a[cd];
        }
    }
}
__global__ void pack(int *a,char *b){
    int i=threadIdx.x;
    if(b[i]>='A' && b[i]<='Z'){
        b[a[i]-1]=b[i];
    }
}

int main()
{
    int n;
    printf("Enter the size of the array:\n");
    // scanf("%d",&n);
    n=8;
    
    int a[n];
    int bc=0;
    printf("Enter the array:\n");
    char b[n] = {'A','B','C','D','e','f','g','H'};
    for(int i = 0; i < n; i++)
    {
        //scanf("%c",&b[i]);
        if(b[i]>='A' && b[i]<='Z'){
            a[i]=1;
            bc++;
        }
        else{
            a[i]=0;
        }
    }
    int *d_a;
    hipMalloc((void **)&d_a,n*sizeof(int));
    hipMemcpy(d_a,a,n*sizeof(int),hipMemcpyHostToDevice);
    char *d_b;
    hipMalloc((void **)&d_b,n*sizeof(char));
    hipMemcpy(d_b,b,n*sizeof(char),hipMemcpyHostToDevice);
    prefixsum<<<1,n>>>(d_a,n,log2(n));
    hipMemcpy(a,d_a,n*sizeof(int),hipMemcpyDeviceToHost);
    printf("The prefix sum of the array is:");
    for(int i=0;i<n;i++)
    {
        printf("%d ",a[i]);
    }
    printf("\n");
    pack<<<1,n>>>(d_a,d_b);
    hipMemcpy(b,d_b,n*sizeof(char),hipMemcpyDeviceToHost);
    printf("The packed array is:");
    for(int i=0;i<bc;i++)
    {
        printf("%c ",b[i]);
    }
}